#include "hip/hip_runtime.h"
#include <cstdio>
#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "matrix.cuh"
#include "device_context.cuh"
#include <tsvd.h>
#include <ctime>
#include <thrust/iterator/counting_iterator.h>
#include<algorithm>

namespace tsvd
{

void row_reverse_trunc_q(const Matrix<float> &Qt, Matrix<float> &QtTrunc, DeviceContext &context){

	auto m = Qt.rows();
	auto k = QtTrunc.rows();
	auto d_q = Qt.data();
	auto d_q_trunc = QtTrunc.data();
	auto counting = thrust::make_counting_iterator <int>(0);
	thrust::for_each(counting, counting+QtTrunc.size(), [=]__device__(int idx){
		int dest_row = idx % k;
		int dest_col = idx / k;
		int src_row = (m - dest_row) - 1;
		int src_col = dest_col;
		float q = d_q[src_col * m + src_row];
		d_q_trunc[idx] = q;
	} );
}

// Calculate U, which is:
// U = A*V/sigma where A is our X Matrix, V is Qt, and sigma is 1/w_i
void calculate_u(const Matrix<float> &X, const Matrix<float> &Qt, const Matrix<float> &w, Matrix<float> &U, DeviceContext &context){

	multiply(X, Qt, U, context, false, true, 1.0f); //A*V
	auto d_u = U.data();
	auto d_sigma = w.data();
	auto column_size = U.rows();
	auto counting = thrust::make_counting_iterator <int>(0);
	thrust::for_each(counting, counting+U.size(), [=]__device__(int idx){
		int column = idx/column_size;
		float sigma = d_sigma[column];
		float u = d_u[idx];
		if(sigma != 0.0){
			d_u[idx] = u * 1.0/sigma;
		} else{
			d_u[idx] = 0.0;
		}
	} );

}

void truncated_svd(const double* _X, double* _Q, double* _w, double* _U, params _param)
{
	try
	{
		//Take in X matrix and allocate for X^TX
		Matrix<float>X(_param.X_m, _param.X_n);
		X.copy(_X);
		Matrix<float>XtX(_param.X_n, _param.X_n);

		//create context
		DeviceContext context;

		//Multiplye X and Xt and output result to XtX
		multiply(X, X, XtX, context, true, false, 1.0f);

		//Set up Q (V^T) and w (singular value) matrices (w is a matrix of size Q.rows() by 1; really just a vector
		Matrix<float>Q(XtX.rows(), XtX.columns()); // n X n -> V^T
		Matrix<float>w(Q.rows(), 1);
		calculate_eigen_pairs_exact(XtX, Q, w, context);

		//Obtain Q^T to obtain vector as row major order
		Matrix<float>Qt(Q.columns(), Q.rows());
		transpose(Q, Qt, context); //Needed for calculate_u()
		Matrix<float>QtTrunc(_param.k, Qt.columns());
		row_reverse_q(Qt, QtTrunc, context);
		Qt.print();
		QtTrunc.print();
		QtTrunc.copy_to_host(_Q); //Send to host

		//Obtain square root of eigenvalues, which are singular values
		w.transform([=]__device__(float elem){
			if(elem > 0.0){
				return std::sqrt(elem);
			}else{
				return 0.0f;
			}
		}
		);

		//Sort from biggest singular value to smallest
		std::vector<double> w_temp(w.size());
		w.copy_to_host(w_temp.data()); //Send to host
		std::reverse(w_temp.begin(), w_temp.end());
		std::copy(w_temp.begin(), w_temp.begin() + _param.k, _w);

		//Get U matrix
		Matrix<float>U(X.columns(), X.columns());
		calculate_u(X, Qt, w, U, context);
		U.copy_to_host(_U); //Send to host

		}
		catch (std::exception e)
		{
			std::cerr << "tsvd error: " << e.what() << "\n";
		}
		catch (std::string e)
		{
			std::cerr << "tsvd error: " << e << "\n";
		}
		catch (...)
		{
			std::cerr << "tsvd error\n";
		}
	}

}
