#include "hip/hip_runtime.h"
#include <cstdio>
#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "matrix.cuh"
#include "device_context.cuh"
#include <tsvd.h>
#include <ctime>
#include <thrust/iterator/counting_iterator.h>
#include<algorithm>

namespace tsvd
{

//Calculate U, which is:
// U = A*V/sigma where A is our X Matrix, V is Qt, and sigma is 1/w_i
void calculate_u(const Matrix<float> &X, const Matrix<float> &Qt, const Matrix<float> &w, Matrix<float> &U, DeviceContext &context){

	multiply(X, Qt, U, context, false, true, 1.0f); //A*V
	auto d_u = U.data();
	auto d_sigma = w.data();
	auto column_size = U.rows();
	auto counting = thrust::make_counting_iterator <int>(0);
	thrust::for_each(counting, counting+U.size(), [=]__device__(int idx){
		int column = idx/column_size;
		float sigma = d_sigma[column];
		float u = d_u[idx];
		if(sigma != 0.0){
			d_u[idx] = u * 1.0/sigma;
		} else{
			d_u[idx] = 0.0;
		}
	} );

}


void truncated_svd(const double* _X, double* _Q, double* _w, double* _U, params _param)
{
	try
	{
		Matrix<float>X(_param.X_m, _param.X_n);
		X.copy(_X);

		Matrix<float>XtX(_param.X_n, _param.X_n);

		//create context
		DeviceContext context;
		multiply(X, X, XtX, context, true, false, 1.0f);

		Matrix<float>Q(XtX.rows(), XtX.columns());
		Matrix<float>w(Q.rows(), 1);

		calculate_eigen_pairs_exact(XtX, Q, w, context);
		Matrix<float>Qt(Q.columns(), Q.rows());
		transpose(Q, Qt, context);
		Qt.copy_to_host(_Q); //Send to host

		w.transform([=]__device__(float elem){
			if(elem > 0.0){
				return std::sqrt(elem);
			}else{
				return 0.0f;
			}
		}
		);
		std::vector<double> w_temp(w.size());
		w.copy_to_host(w_temp.data()); //Send to host
		std::reverse(w_temp.begin(), w_temp.end());
		std::copy(w_temp.begin(), w_temp.begin() + _param.k, _w);

		//Get U matrix
		Matrix<float>U(X.columns(), X.columns());
		calculate_u(X, Qt, w, U, context);
		U.copy_to_host(_U); //Send to host

		}
		catch (std::exception e)
		{
			std::cerr << "tsvd error: " << e.what() << "\n";
		}
		catch (std::string e)
		{
			std::cerr << "tsvd error: " << e << "\n";
		}
		catch (...)
		{
			std::cerr << "tsvd error\n";
		}
	}

}
