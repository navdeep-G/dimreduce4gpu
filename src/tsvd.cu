#include "hip/hip_runtime.h"
#include <cstdio>
#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "matrix.cuh"
#include "device_context.cuh"
#include <tsvd.h>
#include <ctime>


namespace tsvd
{

void truncated_svd(const double* _X, double* _Q, double* _w, params _param)
{
	try
	{
		Matrix<float>X(_param.X_m, _param.X_n);
		X.copy(_X);

		Matrix<float>XtX(_param.X_n, _param.X_n);

		//create context
		DeviceContext context;
		multiply(X, X, XtX, context, true, false, 1.0f);

		Matrix<float>Q(XtX.rows(), XtX.columns());
		Matrix<float>w(Q.rows(), 1);

		calculate_eigen_pairs_exact(XtX, Q, w, context);
		normalize_columns(Q, context);
		Matrix<float>Qt(Q.columns(), Q.rows());
		transpose(Q, Qt, context);
		Qt.print();
		w.transform([=]__device__(float elem){
			if(elem > 0.0){
				return std::sqrt(elem);
			}else{
				return 0.0f;
			}
		}
		);
		w.print();
		}
		catch (std::exception e)
		{
			std::cerr << "tsvd error: " << e.what() << "\n";
		}
		catch (std::string e)
		{
			std::cerr << "tsvd error: " << e << "\n";
		}
		catch (...)
		{
			std::cerr << "tsvd error\n";
		}
	}

}
