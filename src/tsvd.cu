#include "hip/hip_runtime.h"
#include <cstdio>
#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "matrix.cuh"
#include "device_context.cuh"
#include <tsvd.h>
#include <ctime>
#include <thrust/iterator/counting_iterator.h>
#include<algorithm>

namespace tsvd
{

void col_reverse_q(const Matrix<float> &Q, Matrix<float> &QReversed, DeviceContext &context){
	auto n = Q.columns();
	auto m = Q.rows();
	auto k = QReversed.rows();
	auto d_q = Q.data();
	auto d_q_reversed = QReversed.data();
	auto counting = thrust::make_counting_iterator <int>(0);
	thrust::for_each(counting, counting+QReversed.size(), [=]__device__(int idx){
		int dest_row = idx % m;
		int dest_col = idx/m;
		int src_row = dest_row;
		int src_col = (n - dest_col) - 1;
		d_q_reversed[idx] = d_q[src_col * m + src_row];
	} );
}

// Truncated Q to k vectors (truncated svd)
void row_reverse_trunc_q(const Matrix<float> &Qt, Matrix<float> &QtTrunc, DeviceContext &context){

	auto m = Qt.rows();
	auto k = QtTrunc.rows();
	auto d_q = Qt.data();
	auto d_q_trunc = QtTrunc.data();
	auto counting = thrust::make_counting_iterator <int>(0);
	thrust::for_each(counting, counting+QtTrunc.size(), [=]__device__(int idx){
		int dest_row = idx % k;
		int dest_col = idx / k;
		int src_row = (m - dest_row) - 1;
		int src_col = dest_col;
		float q = d_q[src_col * m + src_row];
		d_q_trunc[idx] = q;
	} );
}

// Calculate U, which is:
// U = A*V/sigma where A is our X Matrix, V is Q, and sigma is 1/w_i
void calculate_u(const Matrix<float> &X, const Matrix<float> &Q, const Matrix<float> &w, Matrix<float> &U, DeviceContext &context){

	multiply(X, Q, U, context, false, false, 1.0f); //A*V
	auto d_u = U.data();
	auto d_sigma = w.data();
	auto column_size = U.rows();
	auto counting = thrust::make_counting_iterator <int>(0);
	thrust::for_each(counting, counting+U.size(), [=]__device__(int idx){
		int column = idx/column_size;
		float sigma = d_sigma[column];
		float u = d_u[idx];
		if(sigma != 0.0){
			d_u[idx] = u * 1.0/sigma;
		} else{
			d_u[idx] = 0.0;
		}
	} );

}

void truncated_svd(const double* _X, double* _Q, double* _w, double* _U, params _param)
{
	try
	{
		//Take in X matrix and allocate for X^TX
		Matrix<float>X(_param.X_m, _param.X_n);
		X.copy(_X);
		Matrix<float>XtX(_param.X_n, _param.X_n);

		//create context
		DeviceContext context;

		//Multiplye X and Xt and output result to XtX
		multiply(X, X, XtX, context, true, false, 1.0f);

		//Set up Q (V^T) and w (singular value) matrices (w is a matrix of size Q.rows() by 1; really just a vector
		Matrix<float>Q(XtX.rows(), XtX.columns()); // n X n -> V^T
		Matrix<float>w(Q.rows(), 1);
		calculate_eigen_pairs_exact(XtX, Q, w, context);

		//Obtain Q^T to obtain vector as row major order
		Matrix<float>Qt(Q.columns(), Q.rows());
		transpose(Q, Qt, context); //Needed for calculate_u()
		Matrix<float>QtTrunc(_param.k, Qt.columns());
		row_reverse_trunc_q(Qt, QtTrunc, context);
		QtTrunc.copy_to_host(_Q); //Send to host

		//Obtain square root of eigenvalues, which are singular values
		w.transform([=]__device__(float elem){
			if(elem > 0.0){
				return std::sqrt(elem);
			}else{
				return 0.0f;
			}
		}
		);

		//Sort from biggest singular value to smallest
		std::vector<double> w_temp(w.size());
		w.copy_to_host(w_temp.data()); //Send to host
		std::reverse(w_temp.begin(), w_temp.end());
		std::copy(w_temp.begin(), w_temp.begin() + _param.k, _w);
		Matrix<float>sigma(w.rows(), 1);
		sigma.copy(w_temp.data());

		//Get U matrix
		Matrix<float>U(X.rows(), X.rows());
		Matrix<float>QReversed(Q.rows(), Q.columns());
		col_reverse_q(Q, QReversed, context);
		calculate_u(X, QReversed, sigma, U, context);
		U.copy_to_host(_U); //Send to host

		//Explained variance (WIP)
		Matrix<float>ExplainedVar(w.rows(), 1);
		multiply(U, sigma, ExplainedVar, context, false, false, 1.0f);

		}
		catch (std::exception e)
		{
			std::cerr << "tsvd error: " << e.what() << "\n";
		}
		catch (std::string e)
		{
			std::cerr << "tsvd error: " << e << "\n";
		}
		catch (...)
		{
			std::cerr << "tsvd error\n";
		}
	}

}
