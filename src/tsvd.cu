#include "hip/hip_runtime.h"
#include <cstdio>
#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "matrix.cuh"
#include "device_context.cuh"
#include <tsvd.h>
#include <ctime>
#include <thrust/iterator/counting_iterator.h>
#include<algorithm>

namespace tsvd
{

void divide(const Matrix<float> &XVar, const Matrix<float> &XVarSum, Matrix<float> &ExplainedVarRatio, DeviceContext &context){
	auto d_x_var = XVar.data();
	auto d_x_var_sum = XVarSum.data();
	auto d_expl_var_ratio = ExplainedVarRatio.data();
	auto counting = thrust::make_counting_iterator <int>(0);
	thrust::for_each(counting, counting+ExplainedVarRatio.size(), [=]__device__(int idx){
		float div_val = d_x_var[idx] / d_x_var_sum[0];
		d_expl_var_ratio[idx] = div_val;
	} );
}

void square_val(const Matrix<float> &UmultSigma, Matrix<float> &UmultSigmaSquare, DeviceContext &context){
	auto n = UmultSigma.columns();
	auto m = UmultSigma.rows();
	auto k = UmultSigmaSquare.rows();
	auto d_u_mult_sigma = UmultSigma.data();
	auto d_u_mult_sigma_square = UmultSigmaSquare.data();
	auto counting = thrust::make_counting_iterator <int>(0);
	thrust::for_each(counting, counting+UmultSigmaSquare.size(), [=]__device__(int idx){
		float square_val = std::pow(d_u_mult_sigma[idx],2);
		d_u_mult_sigma_square[idx] = square_val;
	} );
}

void calc_var(const Matrix<float>UmultSigma, Matrix<float> &UmultSigmaVar, int k, DeviceContext &context){
	//Set aside matrix of 1's for getting columnar sums(t(UmultSima) * UmultOnes)
	Matrix<float>UmultOnes(UmultSigma.rows(), 1);
	UmultOnes.fill(1.0f);

	//Allocate matrices for variance calculation
	Matrix<float>UmultSigmaSquare(UmultSigma.rows(), UmultSigma.columns());
	Matrix<float>UmultSigmaSum(k, 1);
	Matrix<float>UmultSigmaSumSquare(k, 1);
	Matrix<float>UmultSigmaSumOfSquare(k, 1);
	Matrix<float>UmultSigmaVarNum(k, 1);

	//Calculate Variance
	square_val(UmultSigma, UmultSigmaSquare, context);
	multiply(UmultSigmaSquare, UmultOnes, UmultSigmaSumOfSquare, context, true, false, 1.0f);
	multiply(UmultSigma, UmultOnes, UmultSigmaSum, context, true, false, 1.0f);
	square_val(UmultSigmaSum, UmultSigmaSumSquare, context);
	//Get rows
	auto m = UmultSigma.rows();
	multiply(UmultSigmaSumOfSquare, m, context);
	subtract(UmultSigmaSumOfSquare, UmultSigmaSumSquare, UmultSigmaVarNum, context);
	auto d_u_sigma_var_num = UmultSigmaVarNum.data();
	auto d_u_sigma_var = UmultSigmaVar.data();
	auto counting = thrust::make_counting_iterator <int>(0);
	thrust::for_each(counting, counting+UmultSigmaVar.size(), [=]__device__(int idx){
		float div_val = d_u_sigma_var_num[idx]/(std::pow(m,2));
		d_u_sigma_var[idx] = div_val;
	} );
}

void col_reverse_q(const Matrix<float> &Q, Matrix<float> &QReversed, DeviceContext &context){
	auto n = Q.columns();
	auto m = Q.rows();
	auto k = QReversed.rows();
	auto d_q = Q.data();
	auto d_q_reversed = QReversed.data();
	auto counting = thrust::make_counting_iterator <int>(0);
	thrust::for_each(counting, counting+QReversed.size(), [=]__device__(int idx){
		int dest_row = idx % m;
		int dest_col = idx/m;
		int src_row = dest_row;
		int src_col = (n - dest_col) - 1;
		d_q_reversed[idx] = d_q[src_col * m + src_row];
	} );
}

// Truncated Q to k vectors (truncated svd)
void row_reverse_trunc_q(const Matrix<float> &Qt, Matrix<float> &QtTrunc, DeviceContext &context){
	auto m = Qt.rows();
	auto k = QtTrunc.rows();
	auto d_q = Qt.data();
	auto d_q_trunc = QtTrunc.data();
	auto counting = thrust::make_counting_iterator <int>(0);
	thrust::for_each(counting, counting+QtTrunc.size(), [=]__device__(int idx){
		int dest_row = idx % k;
		int dest_col = idx / k;
		int src_row = (m - dest_row) - 1;
		int src_col = dest_col;
		float q = d_q[src_col * m + src_row];
		d_q_trunc[idx] = q;
	} );
}

// Calculate U, which is:
// U = A*V/sigma where A is our X Matrix, V is Q, and sigma is 1/w_i
void calculate_u(const Matrix<float> &X, const Matrix<float> &Q, const Matrix<float> &w, Matrix<float> &U, DeviceContext &context){
	multiply(X, Q, U, context, false, false, 1.0f); //A*V
	auto d_u = U.data();
	auto d_sigma = w.data();
	auto column_size = U.rows();
	auto counting = thrust::make_counting_iterator <int>(0);
	thrust::for_each(counting, counting+U.size(), [=]__device__(int idx){
		int column = idx/column_size;
		float sigma = d_sigma[column];
		float u = d_u[idx];
		if(sigma != 0.0){
			d_u[idx] = u * 1.0/sigma;
		} else{
			d_u[idx] = 0.0;
		}
	} );

}

void truncated_svd(const double* _X, double* _Q, double* _w, double* _U, double* _explained_variance, double* _explained_variance_ratio, params _param)
{
	try
	{
		//Take in X matrix and allocate for X^TX
		Matrix<float>X(_param.X_m, _param.X_n);
		X.copy(_X);
		Matrix<float>XtX(_param.X_n, _param.X_n);

		//create context
		DeviceContext context;

		//Multiply X and Xt and output result to XtX
		multiply(X, X, XtX, context, true, false, 1.0f);

		//Set up Q (V^T) and w (singular value) matrices (w is a matrix of size Q.rows() by 1; really just a vector
		Matrix<float>Q(XtX.rows(), XtX.columns()); // n X n -> V^T
		Matrix<float>w(Q.rows(), 1);
		calculate_eigen_pairs_exact(XtX, Q, w, context);

		//Obtain Q^T to obtain vector as row major order
		Matrix<float>Qt(Q.columns(), Q.rows());
		transpose(Q, Qt, context); //Needed for calculate_u()
		Matrix<float>QtTrunc(_param.k, Qt.columns());
		row_reverse_trunc_q(Qt, QtTrunc, context);
		QtTrunc.copy_to_host(_Q); //Send to host

		//Obtain square root of eigenvalues, which are singular values
		w.transform([=]__device__(float elem){
			if(elem > 0.0){
				return std::sqrt(elem);
			}else{
				return 0.0f;
			}
		}
		);

		//Sort from biggest singular value to smallest
		std::vector<double> w_temp(w.size());
		w.copy_to_host(w_temp.data()); //Send to host
		std::reverse(w_temp.begin(), w_temp.end());
		std::copy(w_temp.begin(), w_temp.begin() + _param.k, _w);
		Matrix<float>sigma(_param.k, 1);
		sigma.copy(w_temp.data());

		//Get U matrix
		Matrix<float>U(X.rows(), _param.k);
		Matrix<float>QReversed(Q.rows(), Q.columns());
		col_reverse_q(Q, QReversed, context);
		calculate_u(X, QReversed, sigma, U, context);
		U.copy_to_host(_U); //Send to host

		//Explained Variance
		Matrix<float>UmultSigma(U.rows(), U.columns());
		//U * Sigma
		multiply_diag(U, sigma, UmultSigma, context, false);
		Matrix<float>UmultSigmaVar(_param.k, 1);
		calc_var(UmultSigma, UmultSigmaVar, _param.k, context);
		UmultSigmaVar.copy_to_host(_explained_variance);

		//Explained Variance Ratio
		//Set aside matrix of 1's for getting sum of columnar variances
		Matrix<float>XmultOnes(X.rows(), 1);
		XmultOnes.fill(1.0f);
		Matrix<float>XVar(1, X.columns());
		calc_var(X, XVar, X.columns(), context);
		Matrix<float>XVarSum(1,1);
		multiply(XVar, XmultOnes, XVarSum, context, false, false, 1.0f);
		Matrix<float>ExplainedVarRatio(_param.k, 1);
		divide(UmultSigmaVar, XVarSum, ExplainedVarRatio, context);
		ExplainedVarRatio.copy_to_host(_explained_variance_ratio);

		}
		catch (const std::exception &e)
		{
			std::cerr << "tsvd error: " << e.what() << "\n";
		}
		catch (std::string e)
		{
			std::cerr << "tsvd error: " << e << "\n";
		}
		catch (...)
		{
			std::cerr << "tsvd error\n";
		}
	}

}
