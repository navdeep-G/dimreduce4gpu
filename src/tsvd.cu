#include "hip/hip_runtime.h"
#include <cstdio>
#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "matrix.cuh"
#include "device_context.cuh"
#include <tsvd.h>
#include <ctime>
#include <thrust/iterator/counting_iterator.h>
#include<algorithm>

namespace tsvd
{

void square_val(const Matrix<float> &UmultSigma, Matrix<float> &UmultSigmaSquare, DeviceContext &context){
	auto n = UmultSigma.columns();
	auto m = UmultSigma.rows();
	auto k = UmultSigmaSquare.rows();
	auto d_u_mult_sigma = UmultSigma.data();
	auto d_u_mult_sigma_square = UmultSigmaSquare.data();
	auto counting = thrust::make_counting_iterator <int>(0);
	thrust::for_each(counting, counting+UmultSigmaSquare.size(), [=]__device__(int idx){
		float square_val = std::pow(d_u_mult_sigma[idx],2);
		d_u_mult_sigma_square[idx] = square_val;
	} );
}

void calc_var(Matrix<float> &UmultSigmaSumOfSquare, const Matrix<float> &UmultSigmaSumSquare, Matrix<float> &UmultSigmaVarNum, Matrix<float> &UmultSigmaVar, tsvd_float n, DeviceContext &context){
	multiply(UmultSigmaSumOfSquare, n, context);
	subtract(UmultSigmaSumOfSquare, UmultSigmaSumSquare, UmultSigmaVarNum, context);
	auto d_u_sigma_var_num = UmultSigmaVarNum.data();
	auto d_u_sigma_var = UmultSigmaVar.data();
	auto counting = thrust::make_counting_iterator <int>(0);
	thrust::for_each(counting, counting+UmultSigmaVar.size(), [=]__device__(int idx){
		float div_val = d_u_sigma_var_num[idx]/(std::pow(n,2));
		d_u_sigma_var[idx] = div_val;
	} );
}

void col_reverse_q(const Matrix<float> &Q, Matrix<float> &QReversed, DeviceContext &context){
	auto n = Q.columns();
	auto m = Q.rows();
	auto k = QReversed.rows();
	auto d_q = Q.data();
	auto d_q_reversed = QReversed.data();
	auto counting = thrust::make_counting_iterator <int>(0);
	thrust::for_each(counting, counting+QReversed.size(), [=]__device__(int idx){
		int dest_row = idx % m;
		int dest_col = idx/m;
		int src_row = dest_row;
		int src_col = (n - dest_col) - 1;
		d_q_reversed[idx] = d_q[src_col * m + src_row];
	} );
}

// Truncated Q to k vectors (truncated svd)
void row_reverse_trunc_q(const Matrix<float> &Qt, Matrix<float> &QtTrunc, DeviceContext &context){
	auto m = Qt.rows();
	auto k = QtTrunc.rows();
	auto d_q = Qt.data();
	auto d_q_trunc = QtTrunc.data();
	auto counting = thrust::make_counting_iterator <int>(0);
	thrust::for_each(counting, counting+QtTrunc.size(), [=]__device__(int idx){
		int dest_row = idx % k;
		int dest_col = idx / k;
		int src_row = (m - dest_row) - 1;
		int src_col = dest_col;
		float q = d_q[src_col * m + src_row];
		d_q_trunc[idx] = q;
	} );
}

// Calculate U, which is:
// U = A*V/sigma where A is our X Matrix, V is Q, and sigma is 1/w_i
void calculate_u(const Matrix<float> &X, const Matrix<float> &Q, const Matrix<float> &w, Matrix<float> &U, DeviceContext &context){
	multiply(X, Q, U, context, false, false, 1.0f); //A*V
	auto d_u = U.data();
	auto d_sigma = w.data();
	auto column_size = U.rows();
	auto counting = thrust::make_counting_iterator <int>(0);
	thrust::for_each(counting, counting+U.size(), [=]__device__(int idx){
		int column = idx/column_size;
		float sigma = d_sigma[column];
		float u = d_u[idx];
		if(sigma != 0.0){
			d_u[idx] = u * 1.0/sigma;
		} else{
			d_u[idx] = 0.0;
		}
	} );

}

void truncated_svd(const double* _X, double* _Q, double* _w, double* _U, double* _explained_variance, params _param)
{
	try
	{
		//Take in X matrix and allocate for X^TX
		Matrix<float>X(_param.X_m, _param.X_n);
		X.copy(_X);
		Matrix<float>XtX(_param.X_n, _param.X_n);

		//create context
		DeviceContext context;

		//Multiplye X and Xt and output result to XtX
		multiply(X, X, XtX, context, true, false, 1.0f);

		//Set up Q (V^T) and w (singular value) matrices (w is a matrix of size Q.rows() by 1; really just a vector
		Matrix<float>Q(XtX.rows(), XtX.columns()); // n X n -> V^T
		Matrix<float>w(Q.rows(), 1);
		calculate_eigen_pairs_exact(XtX, Q, w, context);

		//Obtain Q^T to obtain vector as row major order
		Matrix<float>Qt(Q.columns(), Q.rows());
		transpose(Q, Qt, context); //Needed for calculate_u()
		Matrix<float>QtTrunc(_param.k, Qt.columns());
		row_reverse_trunc_q(Qt, QtTrunc, context);
		QtTrunc.copy_to_host(_Q); //Send to host

		//Obtain square root of eigenvalues, which are singular values
		w.transform([=]__device__(float elem){
			if(elem > 0.0){
				return std::sqrt(elem);
			}else{
				return 0.0f;
			}
		}
		);

		//Sort from biggest singular value to smallest
		std::vector<double> w_temp(w.size());
		w.copy_to_host(w_temp.data()); //Send to host
		std::reverse(w_temp.begin(), w_temp.end());
		std::copy(w_temp.begin(), w_temp.begin() + _param.k, _w);
		Matrix<float>sigma(_param.k, 1);
		sigma.copy(w_temp.data());

		//Get U matrix
		Matrix<float>U(X.rows(), _param.k);
		Matrix<float>QReversed(Q.rows(), Q.columns());
		col_reverse_q(Q, QReversed, context);
		calculate_u(X, QReversed, sigma, U, context);
		U.copy_to_host(_U); //Send to host

		//Explained Variance (WIP)
		Matrix<float>UmultSigma(U.rows(), U.columns());
		//U * Sigma
		multiply_diag(U, sigma, UmultSigma, context, false);
		//Set aside matrix of 1's for getting columnar sums(t(UmultSima) * UmultOnes)
		Matrix<float>UmultOnes(UmultSigma.rows(), 1);
		UmultOnes.fill(1.0f);
		//Multiply based on prevous and get sums per column (1st rows is 1st column, etc...)
		Matrix<float>UmultSigmaSquare(UmultSigma.rows(), UmultSigma.columns());
		Matrix<float>UmultSigmaSum(_param.k, 1);
		Matrix<float>UmultSigmaSumSquare(_param.k, 1);
		Matrix<float>UmultSigmaSumOfSquare(_param.k, 1);

		//Calculate Variance
		square_val(UmultSigma, UmultSigmaSquare, context);
		multiply(UmultSigmaSquare, UmultOnes, UmultSigmaSumOfSquare, context, true, false, 1.0f);
		multiply(UmultSigma, UmultOnes, UmultSigmaSum, context, true, false, 1.0f);
		square_val(UmultSigmaSum, UmultSigmaSumSquare, context);
		Matrix<float>UmultSigmaVarNum(_param.k, 1);
		Matrix<float>UmultSigmaVar(_param.k, 1);
		auto m = UmultSigma.rows();
		calc_var(UmultSigmaSumOfSquare, UmultSigmaSumSquare, UmultSigmaVarNum, UmultSigmaVar, m, context);
		UmultSigmaVar.copy_to_host(_explained_variance);

		}
		catch (std::exception e)
		{
			std::cerr << "tsvd error: " << e.what() << "\n";
		}
		catch (std::string e)
		{
			std::cerr << "tsvd error: " << e << "\n";
		}
		catch (...)
		{
			std::cerr << "tsvd error\n";
		}
	}

}
