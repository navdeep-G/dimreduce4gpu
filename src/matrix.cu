#include "hip/hip_runtime.h"
#include "matrix.cuh"
#include "metric.cuh"
#include <algorithm>

namespace scl
{

	void multiply(const Matrix<scl_float>& A, const Matrix<scl_float>& B, Matrix<scl_float>& C, DeviceContext& context, bool transpose_a, bool transpose_b, scl_float alpha)
	{
		hipblasOperation_t op_a = transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
		hipblasOperation_t op_b = transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

		const scl_float beta = 0;

		int m = C.rows();
		int n = C.columns();
		int k = transpose_a ? A.rows() : A.columns();
		int lda = transpose_a ? k : m;
		int ldb = transpose_b ? n : k;
		int ldc = m;

		safe_cublas(hipblasSgemm(context.cublas_handle, op_a, op_b, m, n, k, &alpha, A.data(), lda, B.data(), ldb, &beta, C.data(), ldc));
	}

	void multiply(Matrix<scl_float>& A, const scl_float a, DeviceContext& context)
	{
		thrust::transform(A.dptr(), A.dptr() + A.size(), A.dptr(), [=]__device__ (scl_float val)
		                  {
			                  return val * a;
		                  }
		);
	}

	void subtract(const Matrix<scl_float>& A, const Matrix<scl_float>& B, Matrix<scl_float>& C, DeviceContext& context)
	{
		auto counting = thrust::make_counting_iterator(0);
		const scl_float* d_A = A.data();
		const scl_float* d_B = B.data();
		scl_float* d_C = C.data();
		thrust::for_each(counting, counting + A.rows() * A.columns(), [=]__device__(int idx)
		                 {
			                 d_C[idx] = d_A[idx] - d_B[idx];
		                 });
	}

	void add(const Matrix<scl_float>& A, const Matrix<scl_float>& B, Matrix<scl_float>& C, DeviceContext& context)
	{
		auto counting = thrust::make_counting_iterator(0);
		const scl_float* d_A = A.data();
		const scl_float* d_B = B.data();
		scl_float* d_C = C.data();
		thrust::for_each(counting, counting + A.rows() * A.columns(), [=]__device__(int idx)
		                 {
			                 d_C[idx] = d_A[idx] + d_B[idx];
		                 });
	}

	void transpose(const Matrix<scl_float>& A, Matrix<scl_float>& B, DeviceContext& context)
	{
		scl_check(A.rows() == B.columns()&&A.columns() == B.rows(), "Transpose dimensions incorrect");
		const scl_float alpha = 1.0f;
		const scl_float beta = 0.0f;
		safe_cublas(hipblasSgeam(context.cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, B.rows(), B.columns(), &alpha, A.data(), A.rows(), &beta, NULL, B.rows(), B.data(), B.rows()));
	}

	void linear_solve(const Matrix<scl_float>& A, Matrix<scl_float>& X, const Matrix<scl_float>& B, DeviceContext& context)
	{
		scl_check(A.rows()>= A.columns(),"Linear solve requires m >= n");
		scl_check(X.rows()>= X.columns(),"Linear solve requires n >= k"); //TODO: is this restriction necessary?

		Matrix<scl_float> A_copy(A);
		Matrix<scl_float> B_copy(A.rows(), A.columns());
		thrust::copy(B.dptr(), B.dptr() + B.size(), B_copy.dptr());
		thrust::fill(B_copy.dptr() + B.size(), B_copy.dptr() + B_copy.size(), 0.0f);

		int work_size = 0;
		safe_cusolver(hipsolverDnSgeqrf_bufferSize(context.cusolver_handle, A_copy.rows(), A_copy.columns(), A_copy.data(), A_copy.rows(), &work_size));

		thrust::device_vector<scl_float> work(work_size);
		scl_float* d_work = thrust::raw_pointer_cast(work.data());

		thrust::device_vector<scl_float> tau((std::min)(A.rows(), A.columns()));
		scl_float* d_tau = thrust::raw_pointer_cast(tau.data());

		thrust::device_vector<int> dev_info(1);
		int* d_dev_info = thrust::raw_pointer_cast(dev_info.data());

		safe_cusolver(hipsolverDnSgeqrf(context.cusolver_handle, A_copy.rows(), A_copy.columns(), A_copy.data(), A_copy.rows(), d_tau, d_work, work_size, d_dev_info));

		scl_check(dev_info[0] == 0, "geqrf unsuccessful");

		safe_cusolver(hipsolverDnSormqr(context.cusolver_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, A.rows(), A.columns(), (std::min)(A.rows(), A.columns()), A_copy.data(), A.rows(), d_tau, B_copy.data(), A.rows(), d_work, work_size, d_dev_info));
		scl_check(dev_info[0] == 0, "ormqr unsuccessful");

		Matrix<scl_float> R(A.columns(), A.columns());
		Matrix<scl_float> QTB(A.columns(), B.columns());
		auto counting = thrust::make_counting_iterator(0);
		int n = R.columns();
		int m = A.rows();
		auto d_R = R.data();
		auto d_A_copy = A_copy.data();
		auto d_QTB = QTB.data();
		auto d_B_copy = B_copy.data();
		int qtb_columns = QTB.columns();
		thrust::for_each(counting, counting + R.size(), [=]__device__ (int idx)
		                 {
			                 int row = idx % n;
			                 int column = idx / n;
			                 d_R[idx] = d_A_copy[column * m + row];

			                 if (column < qtb_columns)
			                 {
				                 d_QTB[idx] = d_B_copy[column * m + row];
			                 }
		                 });

		const scl_float alpha = 1.0f;
		safe_cublas(hipblasStrsm(context.cublas_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, QTB.rows(), QTB.columns(), &alpha, R.data(), R.rows(), QTB.data(), QTB.rows()));

		thrust::copy(QTB.dptr(), QTB.dptr() + QTB.size(), X.data());
	}

	void pseudoinverse(const Matrix<scl_float>& A, Matrix<scl_float>& pinvA, DeviceContext& context)
	{
		scl_check(A.rows() == pinvA.columns() && A.columns() == pinvA.rows(), "pseudoinverse dimensions incorrect");

		//Add zero rows if m < n such that m >= n
		Matrix<scl_float> A_extended((std::max)(A.columns(), A.rows()), A.columns());
		auto counting = thrust::make_counting_iterator(0);
		int A_column_size = A.rows();
		int A_extended_column_size = A_extended.rows();
		auto d_A = A.data();
		auto d_A_extended = A_extended.data();

		thrust::for_each(counting, counting + A_extended.size(), [=]__device__(int idx)
		                 {
			                 int row = idx % A_extended_column_size;

			                 if (row < A_column_size)
			                 {
				                 int column = idx / A_extended_column_size;
				                 d_A_extended[idx] = d_A[A_column_size * column + row];
			                 }
			                 else
			                 {
				                 d_A_extended[idx] = 0;
			                 }
		                 });

		int work_size = 0;
		safe_cusolver(hipsolverDnSgesvd_bufferSize(context.cusolver_handle, A_extended.rows(), A_extended.columns(), &work_size));

		Matrix<scl_float> work(work_size, 1);

		Matrix<scl_float> S((std::min)(A_extended.rows(), A_extended.columns()), 1);
		Matrix<scl_float> U(A_extended.rows(), A_extended.rows());
		Matrix<scl_float> VT(A_extended.columns(), A_extended.columns());
		Matrix<int> dev_info(1, 1);

		safe_cusolver (hipsolverDnSgesvd(context.cusolver_handle, 'A', 'A', A_extended.rows(), A_extended.columns(), d_A_extended, A_extended.rows(), S.data(), U.data(), U.rows(), VT.data(), VT.rows(), work.data(), work_size, NULL, dev_info.data()));

		scl_float eps = 1e-5;
		thrust::transform(S.dptr(), S.dptr() + S.size(), S.dptr(), [=]__device__(scl_float val)
		                  {
			                  if (abs(val) < eps)
			                  {
				                  return 0.0;
			                  }
			                  else
			                  {
				                  return 1.0 / val;
			                  }
		                  });

		Matrix<scl_float> UT(A_extended.rows(), A_extended.rows());

		//Calculate transpose of U
		const scl_float alpha = 1.0;
		const scl_float beta = 0.0;
		safe_cublas(hipblasSgeam(context.cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, UT.rows(), UT.columns(), &alpha, U.data(), UT.rows(), &beta,NULL, UT.rows(), UT.data(), UT.rows()));

		safe_cublas(hipblasSdgmm(context.cublas_handle, HIPBLAS_SIDE_LEFT, UT.rows(), UT.columns(), UT.data(), UT.rows(), S.data(), 1, U.data(), U.rows()));

		Matrix<scl_float> pinvA_extended(A_extended.columns(), A_extended.rows());
		multiply(VT, U, pinvA_extended, context, true);

		thrust::copy(pinvA_extended.dptr(), pinvA_extended.dptr() + pinvA.size(), pinvA.dptr());
	}

	void normalize_columns(Matrix<scl_float>& M, Matrix<scl_float>& M_temp, Matrix<scl_float>& column_length, const Matrix<scl_float>& ones, DeviceContext& context)
	{
		thrust::transform(M.dptr(), M.dptr() + M.size(), M_temp.dptr(), sqr_op());
		auto d_column_length = column_length.data();
		auto d_ones = ones.data();
		const scl_float alpha = 1.0f;
		const scl_float beta = 0.0f;
		safe_cublas(hipblasSgemv(context.cublas_handle, HIPBLAS_OP_T, M.rows(), M.columns(), &alpha, M_temp.data(), M.rows(), d_ones, 1, &beta, d_column_length, 1));

		thrust::transform(column_length.dptr(), column_length.dptr() + column_length.size(), column_length.dptr(), [=]__device__(scl_float val)
		                  {
							  if (val == 0.0)
							  {
								  return 0.0;
							  }

			                  return 1.0/ sqrt(val);
		                  });

		safe_cublas(hipblasSdgmm(context.cublas_handle, HIPBLAS_SIDE_RIGHT, M.rows(), M.columns(), M.data(), M.rows(), d_column_length, 1, M.data(), M.rows()));
	}

	void f_normalize(Matrix<scl_float>& M, DeviceContext& context)
	{
		Matrix<scl_float> temp(M.rows(), M.columns());
		thrust::transform(M.dptr(), M.dptr() + M.size(), temp.dptr(), sqr_op());
		scl_float sum = thrust::reduce(temp.dptr(), temp.dptr() + temp.size());
		multiply(M, 1.0 / std::sqrt(sum), context);
		thrust::transform(M.dptr(), M.dptr() + M.size(), temp.dptr(), sqr_op());
		scl_float final_sum = thrust::reduce(temp.dptr(), temp.dptr() + temp.size());
		printf("f norm sum squares: %1.4f\n", final_sum);
	}

	void normalize_columns_cub(Matrix<scl_float>& M, DeviceContext& context)
	{
		//Create alias so device Lamba does not dereference this pointer
		int m = M.rows();

		thrust::device_vector<scl_float> temp(M.size());
		thrust::device_vector<scl_float> length_squared(M.columns());

		thrust::transform(M.dptr(), M.dptr() + M.size(), temp.begin(), [=]__device__(scl_float val)
		                  {
			                  return val * val;
		                  });


		thrust::device_vector<int> column_segments(M.columns() + 1);
		auto counting = thrust::make_counting_iterator(0);
		thrust::transform(counting, counting + column_segments.size(), column_segments.begin(), [=]__device__(int idx)
		                  {
			                  return idx * m;
		                  });

		// Determine temporary device storage requirements
		void* d_temp_storage = NULL;
		size_t temp_storage_bytes = 0;
		auto segments = thrust::raw_pointer_cast(column_segments.data());
		auto sum_in = thrust::raw_pointer_cast(temp.data());
		auto sum_out = thrust::raw_pointer_cast(length_squared.data());
		hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, sum_in, sum_out,
		                                M.columns(), segments, segments + 1);
		// Allocate temporary storage
		hipMalloc(&d_temp_storage, temp_storage_bytes);
		hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, sum_in, sum_out,
		                                M.columns(), segments, segments + 1);

		//Scale
		auto d_length_squared = thrust::raw_pointer_cast(length_squared.data());
		auto d_data = M.data();
		thrust::transform(counting, counting + M.size(), M.dptr(), [=]__device__(int idx)
		                  {
			                  int col = idx / m;

			                  scl_float length_squared = d_length_squared[col];

			                  if (length_squared > 0.0)
			                  {
				                  return d_data[idx] / std::sqrt(d_length_squared[col]);
			                  }
			                  else
			                  {
				                  return 0.0f;
			                  }
		                  });

		hipFree(d_temp_storage);
	}

	void gradient_descent_solve(const Matrix<scl_float>& A, Matrix<scl_float>& X, const Matrix<scl_float>& B, Matrix<scl_float>& R, DeviceContext& context, scl_float eps, scl_float min_rmse_change)
	{
		residual(B, A, X, R, context);

		const int max_iterations = 1000;
		scl_float best_rmse = FLT_MAX;

		for (int i = 0; i < max_iterations; i++)
		{
			scl_float alpha = eps / A.rows();
			const scl_float beta = 1.0f;
			safe_cublas(hipblasSgemm(context.cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, X.rows(), X.columns(), A.rows(), &alpha, A.data(), A.rows(), R.data(), R.rows(), &beta, X.data(), X.rows()));
			//Recalculate residual
			residual(B, A, X, R, context);

			scl_float rmse = rmse_metric(R);

			if (std::abs(best_rmse - rmse) < min_rmse_change)
			{
				break;
			}

			if (rmse > best_rmse)
			{
				eps *= 0.5;
			}
			else
			{
				best_rmse = rmse;
				eps *= 1.05;
			}

		}
	}

	void test_linear_solve()
	{
		DeviceContext context;

		int n = 5;
		int m = 6;
		int k = 3;
		Matrix<scl_float> A(m, n);
		A.random(9);
		Matrix<scl_float> X(n, k);
		X.random(17);
		Matrix<scl_float> B(m, k);
		multiply(A, X, B, context);

		Matrix<scl_float> solution(n, k);
		//linear_solve(A, solution, B, context);
		Matrix<scl_float> R(B.rows(), B.columns());
		gradient_descent_solve(A, solution, B, R, context);
		printf("A\n");
		A.print();
		printf("X\n");
		X.print();
		printf("B\n");
		B.print();
		printf("solution\n");
		solution.print();
	}


	void residual(const Matrix<scl_float>& X, const Matrix<scl_float>& D, const Matrix<scl_float>& S, Matrix<scl_float>& R, DeviceContext& context)
	{
		multiply(D, S, R, context);
		subtract(X, R, R, context);
	}
}
