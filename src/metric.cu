#include "hip/hip_runtime.h"
#include "metric.cuh"
#include "utils.cuh"
#include "matrix.cuh"
#include <thrust/transform_reduce.h>
#include <thrust/device_ptr.h>

namespace scl
{

	scl_float pyksvd_metric(const Matrix<scl_float>& R)
	{
		auto dptr = thrust::device_pointer_cast(R.data());

		scl_float sum_square = thrust::transform_reduce(dptr, dptr + R.size(), sqr_op(),
		                                            0.0, thrust::plus<scl_float>());
		scl_float f_norm = std::sqrt(sum_square);
		return f_norm / R.columns();
	}

	scl_float rmse_metric(const Matrix<scl_float>& R)
	{
		auto dptr = thrust::device_pointer_cast(R.data());

		scl_float MSE = thrust::transform_reduce(dptr, dptr + R.size(), sqr_op(),
		                                     0.0, thrust::plus<scl_float>()) / R.size();
		return std::sqrt(MSE);
	}
}
